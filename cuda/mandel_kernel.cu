#include "hip/hip_runtime.h"
#include <torch/extension.h>

__device__ int mandel_point(double x0, double y0, int max_iter) {
    double x = 0, y = 0;
    int iter = 0;
    while (x*x + y*y <= 4 && iter < max_iter) {
        double x_temp = x*x - y*y + x0;
        y = 2*x*y + y0;
        x = x_temp;
        iter++;
    }
    return iter;
}

__global__ void mandel_kernel(double* x_coords, double* y_coords, int* output,
                            int size, int max_iter) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = mandel_point(x_coords[idx], y_coords[idx], max_iter);
    }
}

torch::Tensor mandelbrot_cuda(torch::Tensor x_coords, torch::Tensor y_coords, int max_iter) {
    auto output = torch::zeros_like(x_coords, torch::kInt32);

    const int threads = 256;
    const int blocks = (x_coords.size(0) + threads - 1) / threads;

    mandel_kernel<<<blocks, threads>>>(
        x_coords.data_ptr<double>(),
        y_coords.data_ptr<double>(),
        output.data_ptr<int>(),
        x_coords.size(0),
        max_iter
    );

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mandelbrot_cuda", &mandelbrot_cuda, "Mandelbrot CUDA kernel");
}
